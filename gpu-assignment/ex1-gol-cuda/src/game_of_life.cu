#include "hip/hip_runtime.h"
#include "common.h"
__global__
void cpu_game_of_life_step(int *current_grid, int *next_grid, int n, int m){
    int neighbours;
    int n_i[8], n_j[8];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < m; j++){
            // count the number of neighbours, clockwise around the current cell.
            neighbours = 0;
            n_i[0] = i - 1; n_j[0] = j - 1;
            n_i[1] = i - 1; n_j[1] = j;
            n_i[2] = i - 1; n_j[2] = j + 1;
            n_i[3] = i;     n_j[3] = j + 1;
            n_i[4] = i + 1; n_j[4] = j + 1;
            n_i[5] = i + 1; n_j[5] = j;
            n_i[6] = i + 1; n_j[6] = j - 1;
            n_i[7] = i;     n_j[7] = j - 1;

            if(n_i[0] >= 0 && n_j[0] >= 0 && current_grid[n_i[0] * m + n_j[0]] == ALIVE) neighbours++;
            if(n_i[1] >= 0 && current_grid[n_i[1] * m + n_j[1]] == ALIVE) neighbours++;
            if(n_i[2] >= 0 && n_j[2] < m && current_grid[n_i[2] * m + n_j[2]] == ALIVE) neighbours++;
            if(n_j[3] < m && current_grid[n_i[3] * m + n_j[3]] == ALIVE) neighbours++;
            if(n_i[4] < n && n_j[4] < m && current_grid[n_i[4] * m + n_j[4]] == ALIVE) neighbours++;
            if(n_i[5] < n && current_grid[n_i[5] * m + n_j[5]] == ALIVE) neighbours++;
            if(n_i[6] < n && n_j[6] >= 0 && current_grid[n_i[6] * m + n_j[6]] == ALIVE) neighbours++;
            if(n_j[7] >= 0 && current_grid[n_i[7] * m + n_j[7]] == ALIVE) neighbours++;

            if(current_grid[i*m + j] == ALIVE && (neighbours == 2 || neighbours == 3)){
                next_grid[i*m + j] = ALIVE;
            } else if(current_grid[i*m + j] == DEAD && neighbours == 3){
                next_grid[i*m + j] = ALIVE;
            }else{
                next_grid[i*m + j] = DEAD;
            }
        }
    }
}


/*
Implements the game of life on a grid of size `n` times `m`, starting from the `initial_state` configuration.

If `nsteps` is positive, returns the last state reached.
*/
__global__
int* cpu_game_of_life(const int *initial_state, int n, int m, int nsteps){
    int *grid = (int *) malloc(sizeof(int) * n * m);
    int *updated_grid = (int *) malloc(sizeof(int) * n * m);
    if(!grid || !updated_grid){
        printf("Error while allocating memory.\n");
        exit(1);
    }
    int current_step = 0;
    int *tmp = NULL;
    memcpy(grid, initial_state, sizeof(int) * n * m);
    while(current_step != nsteps){
        current_step++;
        // Uncomment the following line if you want to print the state at every step
        visualise(VISUAL_ASCII, current_step, grid, n, m);
        cpu_game_of_life_step(grid, updated_grid, n, m);
        // swap current and updated grid
        tmp = grid;
        grid = updated_grid;
        updated_grid = tmp;
    }
    free(updated_grid);
    return grid;
}

#ifndef INCLUDE_CPU_VERSION // do not define the main function if this file is included somewhere else.
int main(int argc, char **argv)
{
    struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
    getinput(argc, argv, opt);
    int n = opt->n, m = opt->m, nsteps = opt->nsteps;
    int *initial_state = (int *) malloc(sizeof(int) * n * m);
    if(!initial_state){
        printf("Error while allocating memory.\n");
        return -1;
    }
    generate_IC(opt->iictype, initial_state, n, m);
    struct timeval start;
    start = init_time();
    int *final_state = cpu_game_of_life(initial_state, n, m, nsteps);
    float elapsed = get_elapsed_time(start);
    printf("Finished GOL in %f ms\n", elapsed);
    free(final_state);
    free(initial_state);
    free(opt);
    return 0;
}
#endif
